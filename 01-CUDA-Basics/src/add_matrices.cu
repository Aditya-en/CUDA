#include <iostream>
#include <hip/hip_runtime.h>
#define N 20

__global__ void Add_matrices(float A[N][N], float B[N][N], float C[N][N])
{
    int i = threadIdx.x;
    int j = threadIdx.y;

    if (i < N && j < N)
    {
        C[i][j] = A[i][j] + B[i][j];
    }
}

int main()
{
    float H_A[N][N], H_B[N][N], H_C[N][N];

    // Initialize the host matrices
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            H_A[i][j] = i + j;
            H_B[i][j] = i - j;
        }
    }

    // Device pointers
    float(*d_A)[N], (*d_B)[N], (*d_C)[N];

    // Allocate memory in device
    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    // Move the matrices to device
    hipMemcpy(d_A, H_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, H_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 blockSize(N, N);
    Add_matrices<<<1, blockSize>>>(d_A, d_B, d_C);

    // Copy the result back to the host
    hipMemcpy(H_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free the device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Print the results
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout << H_C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
