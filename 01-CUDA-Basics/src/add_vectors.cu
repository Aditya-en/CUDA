
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    int N = 100;
    size_t size = N * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the host arrays
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    // Device pointers
    float *d_a, *d_b, *d_c;

    // Allocate memory on the device
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy the data from host to device
    hipMemcpy(d_a, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_B, size, hipMemcpyHostToDevice);

    // Launch the kernel with 1 block and N threads
    add<<<1, N>>>(d_a, d_b, d_c);

    // Copy result back from device to host
    hipMemcpy(h_C, d_c, size, hipMemcpyDeviceToHost);

    // Print the results
    for (int i = 0; i < 10; i++)
    {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
